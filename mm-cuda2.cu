#include "hip/hip_runtime.h"
//----------------------------------------------------------
// Matrix Multiplication - CUDA Version 2 to run on GPUs
//---------------------------------------------------------
//  By Gita Alaghband, Lan Vu 
//  Use shared memory with higher access speed
//  Updated in 8/8/2011
//-----------------------------------------------------------
#include <iostream>
#include <iomanip>
#include <cmath>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>    
#include <hip/hip_runtime.h>

using namespace std;
#define TILE 16

//-----------------------------------------------------------------------
//   Get user input for matrix dimension or printing option
//-----------------------------------------------------------------------
bool GetUserInput(int argc, char *argv[],int& n,int& isPrint)
{
	bool isOK = true;

	if(argc < 2) 
	{
		cout << "Arguments:<X> [<Y>]" << endl;
		cout << "X : Matrix size [X x X]" << endl;
		cout << "Y = 1: print the input/output matrix if X < 10" << endl;
		cout << "Y <> 1 or missing: does not print the input/output matrix" << endl;
		isOK = false;
	}
	else 
	{
		//get matrix size
		n = atoi(argv[1]);
		if (n <= 0) 
		{
			cout << "Matrix size must be larger than 0" <<endl;
			isOK = false;
		}

		//is print the input/output matrix
		if (argc >=3)
			isPrint = (atoi(argv[2])==1 && n <=9)?1:0;
		else
			isPrint = 0;
	}
	return isOK;
}

//-----------------------------------------------------------------------
//Initialize the value of matrix x[n x n]
//-----------------------------------------------------------------------
void InitializeMatrix(float** &x,int n,float value)
{
	x = new float*[n];
	x[0] = new float[n*n];
    srand (time(NULL));

	for (int i = 1; i < n; i++)	x[i] = x[i-1] + n;

	for (int i = 0 ; i < n ; i++)
	{
		for (int j = 0 ; j < n ; j++)
		{
            if (value == 1)  // generate input matrices (a and b)
                x[i][j] = (float)((rand()%10)/(float)2);
            else
                x[i][j] = 0;  // initializing resulting matrix
		}
	}
}

//------------------------------------------------------------------
//Delete matrix x[n x n]
//------------------------------------------------------------------
void DeleteMatrix(float **x,int n)
{
	delete[] x[0];
	delete[] x; 
}

//------------------------------------------------------------------
//Print matrix	
//------------------------------------------------------------------
void PrintMatrix(float **x, int n) 
{
	for (int i = 0 ; i < n ; i++)
	{
		cout<< "Row " << (i+1) << ":\t" ;
		for (int j = 0 ; j < n ; j++)
		{
			printf("%.2f\t", x[i][j]);
		}
		cout<<endl ;
	}
}

//-----------------------------------------------------------------------	
//Do Matrix Multiplication - Version 2: use shared memory
//-----------------------------------------------------------------------	
__global__ void MultiplyMatrix_Version2(float* a, float* b, float* c, int n)
{

	__shared__ float A[TILE][TILE];
	__shared__ float B[TILE][TILE];

	int tx = threadIdx.x;
	int ty = threadIdx.y;
	
	int Row = blockIdx.y*TILE + ty;
	int Col = blockIdx.x*TILE + tx;
	
	float value = 0;

	if (Row < n && Col < n) 
	{
		for (int i = 0; i < n ; i += TILE) 
		{
			// Load the matrices from device memory to shared memory
			// Each thread loads one element of each matrix
			A[ty][tx] = a[ n*Row + (i + tx)]; 
			B[ty][tx] = b[ n*(i + ty) + Col]; 
			// Synchronize to make sure the matrices are loaded
			__syncthreads();	

			// Multiply the two matrices
			// Each thread computes one element of the block sub-matrix
			int m = ((n - i) < TILE)? (n - i): TILE;
			for (int j = 0; j < m; j++)  value += A[ty][j] * B[j][tx];

			// Synchronize to make sure that the preceding computation is done before 
			// loading two new sub-matrices of A and B in the next iteration
			__syncthreads();	
			
		}
		c[Row*n + Col] = value;
	}
}
//------------------------------------------------------------------
// Main Program
//------------------------------------------------------------------
int main(int argc, char *argv[])
{

    float **a, **b,**c; //host pointers
	float *da, *db, *dc; //device pointers
	int n,isPrint;
	double runtime;
	
	if(GetUserInput(argc,argv,n,isPrint)==false) return 1;

    cout << "cuda 2 - gpu matrix multiplication using shared memory " << endl;
    cout << "matrix size is " << n << endl;
	//Initialize the value of matrix a and vetors x, y
	InitializeMatrix(a,n,1.0);
	InitializeMatrix(b,n,1.0);
	InitializeMatrix(c,n,0.0);

	//Print the input matrices
	if (isPrint==1)
	{
		cout<< "Matrix a[n][n]:" << endl;
		PrintMatrix(a,n); 
		cout<< "Matrix b[n][n]:" << endl;
		PrintMatrix(b,n); 
		
	}
	
	runtime = clock()/(float)CLOCKS_PER_SEC;

	//Declare grid size and block size
	int numblock = n/TILE + ((n%TILE)?1:0);
	dim3 dimGrid(numblock,numblock);	
	dim3 dimBlock(TILE,TILE);	

	//Allocate memory on device
	hipMalloc((void**)&da, n*n*sizeof(float));
	hipMalloc((void**)&db, n*n*sizeof(float));
	hipMalloc((void**)&dc, n*n*sizeof(float));

	//Copy data to the device
	hipMemcpy(da, a[0], n*n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b[0], n*n*sizeof(float), hipMemcpyHostToDevice);

	//Do the matrix multiplication on the device (GPU)
	MultiplyMatrix_Version2<<<dimGrid,dimBlock>>>(da,db,dc,n);
	
    hipDeviceSynchronize();

	//Get results from the device
	hipMemcpy(c[0],dc, n*n*sizeof(float),hipMemcpyDeviceToHost);

	runtime = clock() - runtime;

	//The matrix is as below:
	if (isPrint==1)
	{
		cout<< "Matrix c[n][n]:" << endl;
		PrintMatrix(c,n); 
	}

	cout<< "Program runs in " << setiosflags(ios::fixed) << setprecision(8) << (runtime)/float(CLOCKS_PER_SEC) << " seconds\n";

	hipFree(da);
	hipFree(db);
	hipFree(dc);

	DeleteMatrix(a,n);	
	DeleteMatrix(b,n);	
	DeleteMatrix(c,n);	
	
	return 0;
}
